
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
void CuDNNLCNLayer<Dtype,Mtype>::Forward_gpu(const vector<Blob<Dtype,Mtype>*>& bottom,
    const vector<Blob<Dtype,Mtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  MemoryHandler::mallocGPU(&this->tempData1, this->tempDataSize);
  MemoryHandler::mallocGPU(&this->tempData2, this->tempDataSize);

  CUDNN_CHECK(cudnnDivisiveNormalizationForward(
        Caffe::cudnn_handle(), norm_desc_, CUDNN_DIVNORM_PRECOMPUTED_MEANS,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        NULL,  // srcMeansData
        this->tempData1, this->tempData2,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data) );

  MemoryHandler::freeGPU(this->tempData1);
  MemoryHandler::freeGPU(this->tempData2);
}

template <typename Dtype, typename Mtype>
void CuDNNLCNLayer<Dtype,Mtype>::Backward_gpu(const vector<Blob<Dtype,Mtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype,Mtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

  MemoryHandler::mallocGPU(&this->tempData1, this->tempDataSize);
  MemoryHandler::mallocGPU(&this->tempData2, this->tempDataSize);

  CUDNN_CHECK(cudnnDivisiveNormalizationBackward(
        Caffe::cudnn_handle(), norm_desc_, CUDNN_DIVNORM_PRECOMPUTED_MEANS,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        NULL, top_diff,  // NULL - srcMeansData
        this->tempData1, this->tempData2,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff,
        NULL) );

  MemoryHandler::freeGPU(this->tempData1);
  MemoryHandler::freeGPU(this->tempData2);
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNLCNLayer);

}  // namespace caffe
#endif
