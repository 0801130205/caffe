#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <>
void caffe_gpu_gemm<float,float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double,double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemv<float,float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double,double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_axpy<float,float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double,double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float,float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double,double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_axpby<float,float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float,float>(N, beta, Y);
  caffe_gpu_axpy<float,float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double,double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double,double>(N, beta, Y);
  caffe_gpu_axpy<double,double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<float,float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double,double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_asum<float,float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double,double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_scale<float,float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double,double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <typename Dtype, typename Mtype>
__global__ void set_kernel(const int n, const Mtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Mtype>(alpha);
  }
}

template <typename Dtype, typename Mtype>
void caffe_gpu_set(const int N, const Mtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype,Mtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int,int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float,float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double,double>(const int N, const double alpha, double* Y);

template <typename Dtype, typename Mtype>
__global__ void add_scalar_kernel(const int n, const Mtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>(alpha + Get<Mtype>(y[index]));
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype, typename Mtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>( Get<Mtype>(a[index]) + Get<Mtype>(b[index]) );
  }
}

template <>
void caffe_gpu_add<float,float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double,double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype, typename Mtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>( Get<Mtype>(a[index]) - Get<Mtype>(b[index]) );
  }
}

template <>
void caffe_gpu_sub<float,float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double,double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype, typename Mtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>( Get<Mtype>(a[index]) * Get<Mtype>(b[index]) );
  }
}

template <>
void caffe_gpu_mul<float, float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double,double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype, typename Mtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>( Get<Mtype>(a[index]) / Get<Mtype>(b[index]) );
  }
}

template <>
void caffe_gpu_div<float,float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double,double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype, typename Mtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>( abs(Get<Mtype>(a[index])) );
  }
}

template <>
void caffe_gpu_abs<float,float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double,double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype, typename Mtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>( exp(Get<Mtype>(a[index])) );
  }
}

template <>
void caffe_gpu_exp<float,float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double,double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype, typename Mtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Mtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>( pow(Get<Mtype>(a[index]), alpha) );
  }
}

template <>
void caffe_gpu_powx<float,float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double,double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = Get<Dtype>( (Mtype(0) < Get<Mtype>(x[index]))
                                      - (Get<Mtype>(x[index]) < Mtype(0))) );
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = Get<Dtype>( signbit(Get<Mtype>(x[index]))) );

__global__ void popc_kernel(const int n, const float* a,
    const float* b, uint8_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = __popc(static_cast<uint32_t>(a[index]) ^
                      static_cast<uint32_t>(b[index]));
  }
}

__global__ void popcll_kernel(const int n, const double* a,
    const double* b, uint8_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = __popcll(static_cast<uint64_t>(a[index]) ^
                      static_cast<uint64_t>(b[index]));
  }
}

template <>
uint32_t caffe_gpu_hamming_distance<float,float>(const int n, const float* x,
                                  const float* y) {
  // TODO: Fix caffe_gpu_hamming_distance (see failing unit test
  // TestHammingDistanceGPU in test_math_functions.cpp).
  NOT_IMPLEMENTED;
  thrust::device_vector<uint8_t> popcounts(n);
  // NOLINT_NEXT_LINE(whitespace/operators)
  popc_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, thrust::raw_pointer_cast(popcounts.data()));
  return thrust::reduce(popcounts.begin(), popcounts.end(),
                        (uint32_t) 0, thrust::plus<uint32_t>());
}

template <>
uint32_t caffe_gpu_hamming_distance<double,double>(const int n, const double* x,
                                   const double* y) {
  // TODO: Fix caffe_gpu_hamming_distance (see failing unit test
  // TestHammingDistanceGPU in test_math_functions.cpp).
  NOT_IMPLEMENTED;
  thrust::device_vector<uint8_t> popcounts(n);
  // NOLINT_NEXT_LINE(whitespace/operators)
  popcll_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, thrust::raw_pointer_cast(popcounts.data()));
  return thrust::reduce(popcounts.begin(), popcounts.end(),
                        /* NOLINT_NEXT_LINE(build/include_what_you_use) */
                        (uint32_t) 0, thrust::plus<uint32_t>());
}

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float,float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal<float,float>(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar<float,float>(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double,double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal<double,double>(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar<double,double>(n, a, r);
  }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

}  // namespace caffe
